﻿#include <hip/hip_runtime.h>
#include <>

#include <iostream>
using namespace std;

int main()
{
	int count;
	hipGetDeviceCount(&count);
	hipDeviceProp_t prop;
	for (int i = 0; i < count; ++i)
	{
		hipGetDeviceProperties(&prop, i);

		count << "Device" << i << ":" << prop.name << endl;

		count << "Compute capability:" << prop.major << "." << prop.minor << endl;
		count << "Max Grid dimensions: (" <<
			prop.maxGridSize[0] << "x" <<
			prop.maxGridSize[1] << "x" <<
			prop.maxGridSize[2] << "x）" << endl;

		count << "Max block dimensions: (" <<
			prop.maxThreadsDim[0] << "x" <<
			prop.maxThreadsDim[1] << "x" <<
			prop.maxThreadsDim[2] << "x)" << endl;

		getchar()
		return 0;

	}

}