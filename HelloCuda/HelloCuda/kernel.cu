#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__   addArray(int *a, int *b, int *c, int count) {

	int i = threadIdx.x;
	c[i] = a[i] + b[i];

#if 0
	for (int i = 0; i < count; ++i)
		c[i] = a[i] + b[i];
#endif


}

void main() {

	const int count = 5;
	const int size = count * sizeof(int);
	int ha[] = { 1,2,3,4,5};
	int hb[] = { 10,20,30,40,50 };
	int hc[count];

	//cuda var in gpu
	int *da, *db, *dc;
	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size)


		hipMemcpy(da, ha, size, hipMemcpyHostToHost);
		hipMemcpy(db, hb, size, hipMemcpyDeviceToHost);

	addArray <<<1, count >>> (da, db, dc);

	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);
	


	for (int i = 0; i < count; ++i)
	{
		printf("%d", hc[i]);
	}
	getchar();
	
}